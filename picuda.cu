/**
 * calculate pi
 */
#include <stdio.h>
#include <math.h>
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
//Tiempo
#include <ctime>

#define NUMTHREADS 10240
#define ITERATIONS 1e12

/**
 * CUDA Kernel Device code
 * 
 */ 
/*****************************************************************************/

__global__ void calculatePi(double *piTotal, long int iterations, int totalThreads)
{   long int initialIteration, endIteration;
    long int i = 0;
    double piPartial;
    
    //TamanioBloque*IdBloque + IdHilo 
    int index = (blockDim.x * blockIdx.x) + threadIdx.x;

    initialIteration = (iterations/totalThreads) * index;
    endIteration = initialIteration + (iterations/totalThreads) - 1;
    
    i = initialIteration;
    piPartial = 0;
    
    do{
        piPartial = piPartial + (double)(4.0 / ((i*2)+1));
        i++;
        piPartial = piPartial - (double)(4.0 / ((i*2)+1));
        i++;
    }while(i < endIteration);

    piTotal[index] = piPartial;
    
    __syncthreads();
    if(index == 0){
        for(i = 1; i < totalThreads; i++)
            piTotal[0] = piTotal[0] + piTotal[i];
    }
}


/******************************************************************************
 * Host main routine
 */
int main(int argc, char *argv[])
{   
    int totalThreads, blocksPerGrid, threadsPerBlock, i, size;
    long int iterations;
    double *h_pitotal, *d_pitotal;
    
    sscanf(argv[1], "%i", &blocksPerGrid);
    hipError_t err = hipSuccess;

    size = sizeof(double)*NUMTHREADS;
    h_pitotal = (double *)malloc(size);
    if ( h_pitotal == NULL){
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }
    
    for(i = 0; i < NUMTHREADS; i++)
        h_pitotal[i] = 0.0;

    err = hipMalloc((void **)&d_pitotal, size);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    err = hipMemcpy(d_pitotal, h_pitotal, sizeof(double)*NUMTHREADS, hipMemcpyHostToDevice);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    clock_t t;
    t = clock();
    // Lanzar KERNEL
    threadsPerBlock = NUMTHREADS/blocksPerGrid;
    totalThreads = blocksPerGrid * threadsPerBlock;
    iterations = ITERATIONS;
    printf("CUDA kernel launch with %d blocks of %d threads Total: %i\n", blocksPerGrid, threadsPerBlock, totalThreads  );
    calculatePi<<<blocksPerGrid, threadsPerBlock>>>(d_pitotal, iterations, totalThreads);
    err = hipGetLastError();
    if (err != hipSuccess){
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(h_pitotal, d_pitotal, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_pitotal);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("\n%.12f", *h_pitotal);
    // Free host memory
    t = clock() - t;
    printf ("It took me %d clicks (%f seconds).\n",(int)t,((float)t)/CLOCKS_PER_SEC);

    free(h_pitotal);
    err = hipDeviceReset();
    if (err != hipSuccess){
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    return 0;
}

